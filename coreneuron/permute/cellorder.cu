
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA_

//#include <iostream>
//#include <stdio.h>

#include "coreneuron/permute/cellorder.hpp"
#include "coreneuron/network/tnode.hpp"
#include "coreneuron/sim/multicore.hpp"

namespace coreneuron {

#define GPU_A(i)      nt->_actual_a[i]
#define GPU_B(i)      nt->_actual_b[i]
#define GPU_D(i)      nt->_actual_d[i]
#define GPU_RHS(i)    nt->_actual_rhs[i]
#define GPU_PARENT(i) nt->_v_parent_index[i]

__device__
void triang_interleaved2_device(NrnThread* nt, int icore, int ncycle, int* stride, int lastnode)
{
    int icycle = ncycle - 1;
    int istride = stride[icycle];
    int i = lastnode - istride + icore;

    int ip;
    double p;
    while(icycle >= 0) {
        // most efficient if istride equal warpsize, else branch divergence!
        if (icore < istride) {
            ip = GPU_PARENT(i);
            p = GPU_A(i) / GPU_D(i);
            atomicAdd(&GPU_D(ip), - p * GPU_B(i));
            atomicAdd(&GPU_RHS(ip), - p * GPU_RHS(i));
        }
        --icycle;
        istride = stride[icycle];
        i -= istride;
    }
}

__device__
void bksub_interleaved2_device(NrnThread* nt,
                               int root,
                               int lastroot,
                               int icore,
                               int ncycle,
                               int* stride,
                               int firstnode)
{
    for (int i = root; i < lastroot; i += warpsize) {
        GPU_RHS(i) /= GPU_D(i);  // the root
    }

    int i = firstnode + icore;

    int ip;
    for (int icycle = 0; icycle < ncycle; ++icycle) {
        int istride = stride[icycle];
        if (icore < istride) {
            ip = GPU_PARENT(i);
            GPU_RHS(i) -= GPU_B(i) * GPU_RHS(ip);
            GPU_RHS(i) /= GPU_D(i);
        }
        i += istride;
    }
}

__global__
void solve_interleaved2_kernel(NrnThread* nt, InterleaveInfo* ii, int ncore)
{
    int icore = blockDim.x * blockIdx.x + threadIdx.x;
    
    int* ncycles = ii->cellsize;         // nwarp of these
    int* stridedispl = ii->stridedispl;  // nwarp+1 of these
    int* strides = ii->stride;           // sum ncycles of these (bad since ncompart/warpsize)
    int* rootbegin = ii->firstnode;      // nwarp+1 of these
    int* nodebegin = ii->lastnode;       // nwarp+1 of these

    int iwarp = icore / warpsize;     // figure out the >> value
    int ic = icore & (warpsize - 1);  // figure out the & mask
    int ncycle = ncycles[iwarp];
    int* stride = strides + stridedispl[iwarp];
    int root = rootbegin[iwarp];
    int lastroot = rootbegin[iwarp + 1];
    int firstnode = nodebegin[iwarp];
    int lastnode = nodebegin[iwarp + 1];

    triang_interleaved2_device(nt, ic, ncycle, stride, lastnode);
    bksub_interleaved2_device(nt, root + ic, lastroot, ic, ncycle, stride, firstnode);
}

void solve_interleaved2_launcher(NrnThread* nt, InterleaveInfo* info, int ncore)
{
    cudaDeviceSynchronize();
    int threadsPerBlock = warpsize;
    int blocksPerGrid = (ncore + threadsPerBlock - 1) / threadsPerBlock;

    solve_interleaved2_kernel<<<blocksPerGrid,threadsPerBlock>>>(nt, info, ncore);
    cudaDeviceSynchronize();
}

} // namespace coreneuron

#endif // ENABLE_CUDA_
